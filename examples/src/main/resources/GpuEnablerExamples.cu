
#include <hip/hip_runtime.h>

extern "C"
// another simple test kernel
__global__ void multiplyBy2(int *size, int *in, int *out) {
    const int ix = threadIdx.x + blockIdx.x * blockDim.x;

    if (ix < *size) {
        out[ix] = in[ix] * 2;
    }
}


extern "C"
// test reduce kernel that sums elements
__global__ void sum(int *size, int *input, int *output, int *stage, int *totalStages) {
    const long ix = threadIdx.x + blockIdx.x * (long)blockDim.x;
    const int jump = 64 * 256;
    // if (ix == 0) printf("size: %d stage : %d totalStages : %d \n",*size, *stage, *totalStages);
    if (*stage == 0) {
        if (ix < *size) {
            assert(jump == blockDim.x * gridDim.x);
            int result = 0;
            for (long i = ix; i < *size; i += jump) {
                result += input[i];
            }
            input[ix] = result;
        }
    } else if (ix == 0) {
        const long count = (*size < (long)jump) ? *size : (long)jump;
        int result = 0;
        for (long i = 0; i < count; ++i) {
            result += input[i];
        }
        output[0] = result;
    }
}


extern "C"
// test reduce kernel that sums elements
__global__ void sum1(int *size, int *input, int *output) {
    const long ix = threadIdx.x + blockIdx.x * (long)blockDim.x;
    if (ix == 0) {
        int result = 0;
        for (long i = ix; i < *size; i++) {
            result += input[i];
        }
        output[0] = result;
    }
}


extern "C"
__global__ void add(int n, long *a, long *b, long *sum)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i<n)
    {
        sum[i] = a[i] + b[i];
        printf("CUDA KERNEL ADD %ld + %ld = %ld \n",a[i],b[i],sum[i]);
    }

}

extern "C"
__global__ void mul(int n, long *a, long *b, long *sum)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i<n)
    {
        sum[i] = a[i] * b[i];
        printf("CUDA KERNEL MUL %ld * %ld = %ld \n",a[i],b[i],sum[i]);
    }

}

extern "C"
__global__ void arrayTest(int n, long *factor, long *arr,long *result)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i == 0) {
       printf("In ArrayTest n=%d factor=%p arr=%p result=%p \n",n,factor,arr,result);
    }

    if (i<n)
    {
        int idx = i * 3;
        result[idx]=arr[idx] * factor[i];
        result[idx + 1]=arr[idx + 1] * factor[i];
        result[idx + 2]=arr[idx + 2] * factor[i];
        printf("ArrayTest  [%ld] * [%ld %ld %ld] = [%ld %ld %ld] \n", factor[i],
                 arr[idx],arr[idx+1],arr[idx+2],
                result[idx],result[idx+1],result[idx+2]);
    }

}
